#include "hip/hip_runtime.h"
#include "nth_element_kernels.h"

namespace amunmt {
namespace GPU {


#define SHARED_SIZE 512

/////////////////////////////////////////////////////////////////////////////////

#define UNROLL_MAXARG_LOOP_HALF( n, max ) \
  if (tid < (n) && tid + (n) < ( max ) ) { \
    if (sdataHalf[tid + ( n ) ] > sdataHalf[tid]) { \
      sdataHalf[tid] = sdataHalf[tid + ( n ) ]; \
      indices[tid] = indices[tid + ( n ) ]; \
    } \
  }

__global__ void gMaxElement(mblas::MatrixWrapper<NthOut<half>> out,
                            const mblas::MatrixWrapper<half> probsWrap,
                            const mblas::MatrixWrapper<uint> batchPositionWrap,
                            uint numBatches)
{
  extern __shared__ half sdataHalf[];
  __shared__ uint indices[SHARED_SIZE];

  uint tid = threadIdx.x;

  for (uint batchIdx = 0; batchIdx < numBatches; ++batchIdx) {
    uint begin = batchPositionWrap[batchIdx];
    uint end = batchPositionWrap[batchIdx + 1];

    uint i = begin + blockIdx.x * (blockDim.x * 2) + tid;

    sdataHalf[tid] = -65504;

    if (i < end) {
      sdataHalf[tid] = probsWrap[i];
      indices[tid] = i;
    }

    if (i + blockDim.x < end) {
      half a = probsWrap[i];
      half b = probsWrap[i + blockDim.x];
      if (a > b) {
        sdataHalf[tid] = a;
        indices[tid] = i;
      } else {
        sdataHalf[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while (i + 2 * gridDim.x * blockDim.x < end) {
      i += 2 * gridDim.x * blockDim.x;

      half a = probsWrap[i];
      if (a > sdataHalf[tid]) {
        sdataHalf[tid] = a;
        indices[tid] = i;
      }

      if (i + blockDim.x < end) {
        half b = probsWrap[i + blockDim.x];
        if (b > sdataHalf[tid]) {
          sdataHalf[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for (uint s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if (tid < s && tid + s < end) {
        if (sdataHalf[tid + s] > sdataHalf[tid]) {
          sdataHalf[tid] = sdataHalf[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP_HALF(32, end);
    UNROLL_MAXARG_LOOP_HALF(16, end);
    UNROLL_MAXARG_LOOP_HALF(8, end);
    UNROLL_MAXARG_LOOP_HALF(4, end);
    UNROLL_MAXARG_LOOP_HALF(2, end);
    UNROLL_MAXARG_LOOP_HALF(1, end);

    if (tid == 0) {
      out[blockIdx.x + batchIdx * gridDim.x] = {indices[0], sdataHalf[0]};
    }
    __syncthreads();
  }
}

__global__ void gMaxElementUpdate(mblas::MatrixWrapper<NthOut<half>> out,
                                  mblas::MatrixWrapper<half> probsWrap,
                                  mblas::MatrixWrapper<NthOut<float>> resNewWrap,
                                  const mblas::MatrixWrapper<uint> batchPositionWrap,
                                  const mblas::MatrixWrapper<uint> cumBeamSizesWrap,
                                  uint numBlocks)
{
  extern __shared__ half sdataHalf[];
  __shared__ uint indices[SHARED_SIZE];
  __shared__ half bestBinCostHalf;
  __shared__ uint bestBinCostIdx;

  const uint tid = threadIdx.x;
  const uint batchIdx = blockIdx.x;
  const uint N = batchPositionWrap[batchIdx + 1] - batchPositionWrap[batchIdx];
  uint num_bins = uint(N / (2 * SHARED_SIZE)) + uint(N % (2 * SHARED_SIZE) != 0);
  //if (num_bins > 500) {
  //  num_bins = 500;
  //}

  for (uint pos = cumBeamSizesWrap[batchIdx]; pos < cumBeamSizesWrap[batchIdx + 1]; ++pos) {
    uint i = tid;

    sdataHalf[tid] = -65504;

    if (i < num_bins) {
      sdataHalf[tid] = out[batchIdx * numBlocks + i].score;
      indices[tid] = i;
    }

    if (i + blockDim.x < num_bins) {
      half a = out[batchIdx * numBlocks + i].score;
      half b = out[batchIdx * numBlocks + i + blockDim.x].score;
      if (a > b) {
        sdataHalf[tid] = a;
        indices[tid] = i;
      } else {
        sdataHalf[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while (i + 2 * blockDim.x < num_bins) {
      i += 2 * blockDim.x;

      half a = out[batchIdx * numBlocks + i].score;
      if (a > sdataHalf[tid]) {
        sdataHalf[tid] = a;
        indices[tid] = i;
      }

      if (i + blockDim.x < num_bins) {
        half b = out[batchIdx * numBlocks + i + blockDim.x].score;
        if (b > sdataHalf[tid]) {
          sdataHalf[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for (uint s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if (tid < s && tid + s < num_bins) {
        if (sdataHalf[tid + s] > sdataHalf[tid]) {
          sdataHalf[tid] = sdataHalf[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP_HALF(32, num_bins);
    UNROLL_MAXARG_LOOP_HALF(16, num_bins);
    UNROLL_MAXARG_LOOP_HALF(8, num_bins);
    UNROLL_MAXARG_LOOP_HALF(4, num_bins);
    UNROLL_MAXARG_LOOP_HALF(2, num_bins);
    UNROLL_MAXARG_LOOP_HALF(1, num_bins);

    if (tid == 0) {
      bestBinCostHalf = sdataHalf[0];
      bestBinCostIdx = batchIdx * numBlocks + indices[0];

      probsWrap[ out[bestBinCostIdx].ind ] = -65504;

      resNewWrap[pos].ind = out[bestBinCostIdx].ind;
      resNewWrap[pos].score = bestBinCostHalf;
    }

    __syncthreads();

    i = batchPositionWrap[batchIdx] + (bestBinCostIdx - batchIdx * numBlocks) * (blockDim.x * 2) + tid;
    const uint dist = num_bins * 2 * blockDim.x;

    sdataHalf[tid] = -65504;

    if (i < batchPositionWrap[batchIdx + 1]) {
      sdataHalf[tid] = probsWrap[i];
      indices[tid] = i;
    }

    if (i + blockDim.x < batchPositionWrap[batchIdx + 1]) {
      half a = probsWrap[i];
      half b = probsWrap[i+blockDim.x];
      if (a > b) {
        sdataHalf[tid] = a;
        indices[tid] = i;
      } else {
        sdataHalf[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while (i + dist < batchPositionWrap[batchIdx + 1]) {
      i += dist;

      half a = probsWrap[i];
      if (a > sdataHalf[tid]) {
        sdataHalf[tid] = a;
        indices[tid] = i;
      }

      if (i + blockDim.x < batchPositionWrap[batchIdx + 1]) {
        half b = probsWrap[i + blockDim.x];
        if (b > sdataHalf[tid]) {
          sdataHalf[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for (uint s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if (tid < s && tid + s < batchPositionWrap[batchIdx + 1]) {
        if (sdataHalf[tid + s] > sdataHalf[tid]) {
          sdataHalf[tid] = sdataHalf[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP_HALF(32, batchPositionWrap[batchIdx + 1]);
    UNROLL_MAXARG_LOOP_HALF(16, batchPositionWrap[batchIdx + 1]);
    UNROLL_MAXARG_LOOP_HALF(8, batchPositionWrap[batchIdx + 1]);
    UNROLL_MAXARG_LOOP_HALF(4, batchPositionWrap[batchIdx + 1]);
    UNROLL_MAXARG_LOOP_HALF(2, batchPositionWrap[batchIdx + 1]);
    UNROLL_MAXARG_LOOP_HALF(1, batchPositionWrap[batchIdx + 1]);

    if (tid == 0) {
      out[bestBinCostIdx] = {indices[0], sdataHalf[0]};
    }
    __syncthreads();
  }
}

__global__ void gGetValueByKey(mblas::MatrixWrapper<half> out,
                              const   mblas::MatrixWrapper<half> in,
                              uint* indices, uint n)
{
  uint tid = threadIdx.x  + blockDim.x * blockIdx.x;
  if (tid < n) {
    uint index = indices[tid];
    out[tid] = in[index];
  }
}

}
}

