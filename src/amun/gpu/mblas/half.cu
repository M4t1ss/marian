#include "hip/hip_runtime.h"
#include <stdint.h>
#include "half.h"

/*
  Copyright (c) 2015, Norbert Juffa
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions
  are met:

  1. Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

  2. Redistributions in binary form must reproduce the above copyright
     notice, this list of conditions and the following disclaimer in the
     documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
  LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

__fp16 uint16_as_fp16 (uint16_t a)
{
    __fp16 res;
#if defined (__cplusplus)
    memcpy (&res, &a, sizeof (res));
#else /* __cplusplus */
    volatile union {
        __fp16 f;
        uint16_t i;
    } cvt;
    cvt.i = a;
    res = cvt.f;
#endif /* __cplusplus */
    return res;
}

uint32_t fp32_as_uint32 (float a)
{
    uint32_t res;
#if defined (__cplusplus)
    memcpy (&res, &a, sizeof (res));
#else /* __cplusplus */
    volatile union {
        float f;
        uint32_t i;
    } cvt;
    cvt.f = a;
    res = cvt.i;
#endif /* __cplusplus */
    return res;
}

/* host version of device function __float2half_rn() */
__fp16 float2half_rn (float a)
{
    uint32_t ia = fp32_as_uint32 (a);
    uint16_t ir;

    ir = (ia >> 16) & 0x8000;
    if ((ia & 0x7f800000) == 0x7f800000) {
        if ((ia & 0x7fffffff) == 0x7f800000) {
            ir |= 0x7c00; /* infinity */
        } else {
            ir = 0x7fff; /* canonical NaN */
        }
    } else if ((ia & 0x7f800000) >= 0x33000000) {
        int shift = (int)((ia >> 23) & 0xff) - 127;
        if (shift > 15) {
            ir |= 0x7c00; /* infinity */
        } else {
            ia = (ia & 0x007fffff) | 0x00800000; /* extract mantissa */
            if (shift < -14) { /* denormal */
                ir |= ia >> (-1 - shift);
                ia = ia << (32 - (-1 - shift));
            } else { /* normal */
                ir |= ia >> (24 - 11);
                ia = ia << (32 - (24 - 11));
                ir = ir + ((14 + shift) << 10);
            }
            /* IEEE-754 round to nearest of even */
            if ((ia > 0x80000000) || ((ia == 0x80000000) && (ir & 1))) {
                ir++;
            }
        }
    }
    return uint16_as_fp16 (ir);
}
