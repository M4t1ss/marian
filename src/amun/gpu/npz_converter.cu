#include "hip/hip_runtime.h"
#include "npz_converter.h"
#include "common/exception.h"

using namespace std;

namespace amunmt {
namespace GPU {

NpzConverter::NpyMatrixWrapper::NpyMatrixWrapper(const cnpy::NpyArray& npy)
: npy_(npy)
{
  string before = Debug();
  Clip(-1, 1);
  string after = Debug();
  cerr << "before=" << before << " after=" << after << endl;
}

void NpzConverter::NpyMatrixWrapper::Clip(float minVal, float maxVal)
{
  float *d = data();
  size_t size = size1() * size2();
  for (size_t i = 0; i < size; ++i) {
    float &val = d[i];
    if (val < minVal) {
      val = minVal;
    }
    else if (val > maxVal) {
      val = maxVal;
    }
  }
}

std::string NpzConverter::NpyMatrixWrapper::Debug(size_t verbosity) const
{
  float min = 2423432;
  float max = -454534534;
  float *d = data();
  size_t size = size1() * size2();
  for (size_t i = 0; i < size; ++i) {
    float val = d[i];
    if (val < min) {
      min = val;
    }
    if (val > max) {
      max = val;
    }
  }

  std::stringstream strm;
  strm << "min/max=" << min << "/"  << max;
  return strm.str();
}

NpzConverter::NpzConverter(const std::string& file)
  : model_(cnpy::npz_load(file)),
    destructed_(false)
{
}

NpzConverter::~NpzConverter() {
  if(!destructed_)
    model_.destruct();
}

void NpzConverter::Destruct() {
  model_.destruct();
  destructed_ = true;
}

std::shared_ptr<mblas::Matrix> NpzConverter::get(const std::string& key, bool mandatory, bool transpose) const
{
  std::shared_ptr<mblas::Matrix> ret;
  auto it = model_.find(key);
  if(it != model_.end()) {
    NpyMatrixWrapper np(it->second);
    mblas::Matrix *matrix = new mblas::Matrix(np.size1(), np.size2(), 1, 1);
    mblas::copy(np.data(), np.size(), matrix->data(), hipMemcpyHostToDevice);

    if (transpose) {
      mblas::Transpose(*matrix);
    }

    ret.reset(matrix);
  }
  else if (mandatory) {
    std::cerr << "Error: Matrix not found:" << key << std::endl;
    //amunmt_UTIL_THROW2(strm.str()); //  << key << std::endl
    abort();
  }
  else {
    mblas::Matrix *matrix = new mblas::Matrix();
    ret.reset(matrix);
  }

  //std::cerr << "key=" << key << " " << matrix.Debug(1) << std::endl;
  return ret;
}

std::shared_ptr<mblas::Matrix> NpzConverter::getFirstOfMany(const std::vector<std::pair<std::string, bool>> keys, bool mandatory) const
{
  std::shared_ptr<mblas::Matrix> ret;
  for (auto key : keys) {
    auto it = model_.find(key.first);
    if(it != model_.end()) {
      NpyMatrixWrapper np(it->second);
      mblas::Matrix *matrix = new mblas::Matrix(np.size1(), np.size2(), 1, 1);
      mblas::copy(np.data(), np.size(), matrix->data(), hipMemcpyHostToDevice);

      if (key.second) {
        mblas::Transpose(*matrix);
      }
      ret.reset(matrix);
      return ret;
    }
  }

  if (mandatory) {
    std::cerr << "Error: Matrix not found:" << keys[0].first << std::endl;
    //amunmt_UTIL_THROW2(strm.str()); //  << key << std::endl
    abort();
  }
  else {
    std::cerr << "Optional matrix not found, continuing: " << keys[0].first << std::endl;
  }

  return ret;

}


}
}
