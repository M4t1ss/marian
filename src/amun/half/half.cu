
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hip/hip_fp16.h>
#include <chrono>

/////////////////////////////////////////////////////////////////////////////

void GPU_fill_rand(half *A, int nr_rows_A, int nr_cols_A) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with random numbers on the device
     /* curandGenerateUniform(prng, A, nr_rows_A * nr_cols_A); */
}

/////////////////////////////////////////////////////////////////////////////

void gpu_blas_mmul(const half *A, const half *B, half *C, const int m, const int k, const int n) {
     int lda=m,ldb=k,ldc=m;

     half alf_h;
     half *alpha_h = &alf_h;

     half bet_h;
     half *beta_h = &bet_h;

     // Create a handle for CUBLAS
     hipblasHandle_t handle;
     hipblasCreate(&handle);


     // Do the actual multiplication
     for (size_t i = 0; i < 1; ++i) {
        hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_h,
            A, lda, B, ldb, beta_h, C, ldc);
     }
     hipStreamSynchronize(0);

     // Destroy the handle
     hipblasDestroy(handle);
}

/////////////////////////////////////////////////////////////////////////////

__global__ void gPlusTanh(const half *A, const half *B, half *C, size_t size)
{
  int i = threadIdx.x  + blockDim.x * blockIdx.x;
  if (i < size) {
    half res = A[i] + B[i];
    //res = tanh(res);
    C[i] = res; 
  }
}

/////////////////////////////////////////////////////////////////////////////

int main() {
    std::chrono::time_point<std::chrono::system_clock> start, end1, end2;
    start = std::chrono::system_clock::now();

     // Allocate 3 arrays on CPU
     int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

     // for simplicity we are going to use square arrays
     nr_rows_A = 512;
     nr_cols_A = 512;
     nr_rows_B = 512;
     nr_cols_B = 85000;
     nr_rows_C = 520;
     nr_cols_C = 85000;

     // Allocate 3 arrays on GPU
     half *d_A, *d_B, *d_C;
     hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(half));
     hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(half));
     hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(half));

     for (size_t i = 0; i < 1000; ++i) {
		 // Fill the arrays A and B on GPU with random numbers
		 GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
		 GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

		 // Multiply A and B on GPU
		 gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
     }

     // Copy (and print) the result on host memory

     //Free GPU memory
     hipFree(d_A);
     hipFree(d_B);
     hipFree(d_C);  

     end1 = std::chrono::system_clock::now();
     std::chrono::duration<double> elapsed1 = end1 - start;
     std::cout << "multiplication: " << elapsed1.count() << "s\n";

     // element-wise tanh(x+y)
     nr_rows_A = 520;
     nr_cols_A = 85000;
     nr_rows_B = 520;
     nr_cols_B = 85000;
     int size = nr_rows_A * nr_cols_A;

     hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
     hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
     hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

     size_t threads = 512;
     size_t blocks =  (size / threads) + ((size % threads == 0) ?  0 : 1);

     GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
     GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

     for (size_t i = 0; i < 1000; ++i) {
       gPlusTanh<<<blocks, threads>>>(d_A, d_B, d_C, size);
     }
     hipStreamSynchronize(0);

     //Free GPU memory
     hipFree(d_A);
     hipFree(d_B);
     hipFree(d_C);  

     end2 = std::chrono::system_clock::now();
     std::chrono::duration<double> elapsed2 = end2 - end1;
     std::cout << "element-wise tanh(x+y): " << elapsed2.count() << "s\n";

     return 0;
 }
